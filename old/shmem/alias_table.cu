#include "hip/hip_runtime.h"
#include "alias_table.cuh"
// template <typename T>
__global__ void shmem_kernel(int *ids, float *weights, size_t size, size_t num,
                             Vector<int> out) {

  __shared__ alias_table_shmem<int> tables[WARP_PER_SM];
  alias_table_shmem<int> *table = &tables[WID];
  // printf("table size %llu\n",table->size);

  table->Init();
  if (LID == 0) {
    printf("table large size %llu\n", table->large.capacity);
  }
  if (TID == 0) {
    printf("load\n");
  }
  table->load(ids, weights, size);
  if (TID == 0) {
    printf("construct\n");
  }
  table->construct();
  if (TID == 0) {
    printf("roll\n");
  }
  table->roll_atomic(out, num);
  if (LID == 0) {
      printf("out: ");
      printD(out.data, out.Size());
  }
}

__global__ void shmem_kernel(int *ids, float *weights, size_t size, size_t num,
                             int * out) {

  __shared__ alias_table_shmem<int> tables[WARP_PER_SM];
  alias_table_shmem<int> *table = &tables[WID];
  // printf("table size %llu\n",table->size);

  table->Init();
  if (LID == 0) {
    printf("table large size %llu\n", table->large.capacity);
  }
  if (TID == 0) {
    printf("load\n");
  }
  table->load(ids, weights, size);
  if (TID == 0) {
    printf("construct\n");
  }
  table->construct();
  if (TID == 0) {
    printf("roll\n");
  }
  table->roll_atomic(out, num);
  if (LID == 0) {
      printf("out: ");
      printD(out, num);
  }
}