#include "hip/hip_runtime.h"
#include "sampler.cuh"
#include "alias_table.cuh"
#include "util.cuh"
#define paster(n) printf("var: " #n " =  %d\n", n)

__global__ void sample_kernel_ptr(Sampler *sampler)
{
  __shared__ alias_table_shmem<uint32_t> tables[WARP_PER_SM];
  alias_table_shmem<uint32_t> *table = &tables[WID];
  int wid = WID;
  sample_result &result = sampler->result;
  gpu_graph &ggraph = sampler->ggraph;

  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  // if (TID == 0)
  //   printf("%s\t %s :%d\n", __FILE__, __PRETTY_FUNCTION__, __LINE__);

  // bool finished; //notFinish
  __shared__ uint32_t current_itr;
  if (threadIdx.x == 0)
    current_itr = 0;
  __syncthreads();
  for (; current_itr < result.hop_num - 1;)
  {
    // if (TID == 0)
    //   printf("==================== start itr %d ================================\n\n", current_itr);
    sample_job job;
    if (LID == 0)
      job = result.requireOneJob(current_itr);
    uint32_t idx = __shfl_sync(0xffffffff, job.idx, 0);
    bool val = __shfl_sync(0xffffffff, job.val, 0);
    uint32_t node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    while (val)
    {
#ifdef check
      if (LID == 0)
        printf("GWID %d itr %d got one job idx %u node_id %u with degree %d \n", GWID, current_itr, idx, node_id, ggraph.getDegree(node_id));
#endif
      // shuffle id
      // table->Init();
      // paster(ggraph.getDegree(node_id));
      table->loadFromGraph(ggraph.getNeighborPtr(node_id), ggraph, ggraph.getDegree(node_id), current_itr);
      // printf("load done\n");
      table->construct();
      // printf("construct done\n");
      uint32_t target_size = MIN(ggraph.getDegree(node_id), result.hops[current_itr + 1]);
      if (target_size > ELE_PER_WARP && LID == 0)
        printf("high degree %d potential overflow \n", target_size);
      // if (ggraph.getDegree(node_id)>16&& LID == 0)
      //   printf(" degree %d  \n", ggraph.getDegree(node_id));
      // paster(result.getAddr(current_itr));
      table->roll_atomic(result.getNextAddr(current_itr), target_size, &state, result); //(T *array, int count, hiprandState *state, sample_result job)
      if (LID == 0)
        job = result.requireOneJob(current_itr);
      idx = __shfl_sync(0xffffffff, job.idx, 0);
      val = __shfl_sync(0xffffffff, job.val, 0);
      node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    }
    // TODO
    // active_size(__LINE__);
    __syncthreads();
    if (threadIdx.x == 0)
    {
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

__global__ void init_kernel_ptr(Sampler *sampler)
{
  if (TID == 0)
  {
    sampler->result.setAddrOffset();
    // printD(sampler->result.addr_offset, 3);
    // printD(sampler->result.hops, 3);
    // printD(sampler->result.job_sizes, 3);
    // printD(sampler.result.job_sizes_h, 2);
    // printD(sampler.result.addr_offset, 2);
  }
}
__global__ void print_result(Sampler *sampler)
{
  if (TID == 0)
  {
    printf("result: \n");
    printD(sampler->result.data, sampler->result.capacity);
  }
}
void Start(Sampler sampler)
{
  printf("%s\t %s :%d\n", __FILE__, __PRETTY_FUNCTION__, __LINE__);

  int device;
  hipDeviceProp_t prop;
  // int activeWarps;
  // int maxWarps;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;
  paster(n_sm);

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  H_ERR(hipMemcpy(sampler_ptr, &sampler,
                   sizeof(Sampler), hipMemcpyHostToDevice));

  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);
  sample_kernel_ptr<<<n_sm, 256, 0, 0>>>(sampler_ptr);
#ifdef check
  print_result<<<1, 32, 0, 0>>>(sampler_ptr);
#endif
  HERR(hipDeviceSynchronize());
  HERR(hipPeekAtLastError());
}

// printf("------------------TB %d go to itr %d\n", TBID, current_itr);
// printf("----------------NextItr wl size %d\n", result.job_sizes[current_itr + 1]);
// printD(result.job_sizes, 3);
// for (size_t i = 0; i < result.job_sizes[current_itr]; i++)
// {
//   printf("%d \t", result.getNodeId(i, current_itr));
// }
// printf("\n");
// printD(result.data, result.capacity);

// __global__ void sample_kernel(Sampler sampler)
// {
//   __shared__ alias_table_shmem<uint32_t> tables[WARP_PER_SM];
//   alias_table_shmem<uint32_t> *table = &tables[WID];
//   int wid = WID;

//   hiprandState state;
//   hiprand_init(TID, 0, 0, &state);

//   // if (TID == 0)
//   //   printf("%s\t %s :%d\n", __FILE__, __PRETTY_FUNCTION__, __LINE__);

//   // bool finished; //notFinish
//   for (; sampler.result.current_itr < sampler.result.hop_num;)
//   {
//     sample_job job;
//     if (LID == 0)
//       job = sampler.result.requireOneJob();
//     uint32_t idx = __shfl_sync(0xffffffff, job.idx, 0);
//     bool val = __shfl_sync(0xffffffff, job.val, 0);
//     uint32_t node_id = __shfl_sync(0xffffffff, job.node_id, 0);
//     while (val)
//     {
//       if (LID == 0)
//         printf("GWID %d got one job idx %u id %u\n", GWID, idx, node_id);
//       // shuffle id
//       // table->Init();
//       table->loadFromGraph(sampler.ggraph.getNeighborPtr(node_id), sampler.ggraph, sampler.ggraph.getDegree(node_id));
//       table->construct();
//       uint32_t target_size = MIN(sampler.ggraph.getDegree(node_id), sampler.result.hops[sampler.result.current_itr + 1]);
//       if (target_size > 0)
//         table->roll_atomic(sampler.result.getAddr(idx, sampler.result.current_itr), target_size, &state, sampler.result); //(T *array, int count, hiprandState *state, sample_result job)
//       if (LID == 0)
//         job = sampler.result.requireOneJob();
//       // __syncwarp(0xffffffff);
//       idx = __shfl_sync(0xffffffff, job.idx, 0);
//       val = __shfl_sync(0xffffffff, job.val, 0);
//       node_id = __shfl_sync(0xffffffff, job.node_id, 0);
//     }
//     if (threadIdx.x == 0)
//       sampler.result.NextItr();
//     __syncthreads();
//   }
// }

// __global__ void init_kernel(Sampler sampler)
// {
//   if (TID == 0)
//   {
//     sampler.result.setAddrOffset();
//     printD(sampler.result.addr_offset, 3);
//     printD(sampler.result.hops, 3);
//     printD(sampler.result.job_sizes, 3);
//     // printD(sampler.result.job_sizes_h, 2);
//     // printD(sampler.result.addr_offset, 2);
//   }
// }
