#include "hip/hip_runtime.h"
#include "sampler.cuh"
#include "alias_table.cuh"
#include "util.cuh"
#define paster(n) printf("var: " #n " =  %d\n", n)

__device__ void SampleUsingShmem(sample_result &result, gpu_graph *ggraph, alias_table_shmem<uint> *table, hiprandState state, int current_itr, int idx, int node_id)
{

#ifdef check
  if (LID == 0)
    printf("GWID %d itr %d got one job idx %u node_id %u with degree %d \n", GWID, current_itr, idx, node_id, ggraph->getDegree(node_id));
#endif
  bool not_all_zero = table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph, ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero)
  {
    table->construct();
    uint target_size = MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    if (target_size > ELE_PER_WARP && LID == 0)
      printf("high degree %d potential overflow \n", target_size);
    table->roll_atomic(result.getNextAddr(current_itr), target_size, &state, result);
  }
  table->Clean();
}

__global__ void sample_kernel(Sampler *sampler)
{
  sample_result &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0)
    current_itr = 0;
  __syncthreads();
  for (; current_itr < result.hop_num - 1;)
  {
    sample_job job;
    if (LID == 0)
      job = result.requireOneJob(current_itr);
    __syncwarp(0xffffffff);
    job.idx = __shfl_sync(0xffffffff, job.idx, 0);
    job.val = __shfl_sync(0xffffffff, job.val, 0);
    job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    if (job.val)
    {
      if (ggraph->getDegree(job.node_id) < ELE_PER_WARP)
      {
        __shared__ alias_table_shmem<uint> tables[WARP_PER_SM];
        alias_table_shmem<uint> *table = &tables[WID];
        SampleUsingShmem(result, ggraph, table, state, current_itr, job.idx, job.node_id);
        if (LID == 0)
          job = result.requireOneJob(current_itr);
        job.idx = __shfl_sync(0xffffffff, job.idx, 0);
        job.val = __shfl_sync(0xffffffff, job.val, 0);
        job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
      }
      else
      {
        if (LID == 0)
          printf("need larger buf for id %d degree %d \n", job.node_id, ggraph->getDegree(job.node_id));
      }
    }
    __syncthreads();
    if (threadIdx.x == 0)
    {
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

__global__ void init_kernel_ptr(Sampler *sampler)
{
  if (TID == 0)
  {
    sampler->result.setAddrOffset();
  }
}
__global__ void print_result(Sampler *sampler)
{
  if (TID == 0)
  {
    printf("result: \n");
    printD(sampler->result.data, sampler->result.capacity);
  }
}
void Start(Sampler sampler)
{
  // printf("%s\t %s :%d\n", __FILE__, __PRETTY_FUNCTION__, __LINE__);
  printf("ELE_PER_WARP %d\n ", ELE_PER_WARP);

  int device;
  hipDeviceProp_t prop;
  // int activeWarps;
  // int maxWarps;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;
  paster(n_sm);

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  H_ERR(hipMemcpy(sampler_ptr, &sampler,
                   sizeof(Sampler), hipMemcpyHostToDevice));

  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);
  sample_kernel<<<n_sm, 256, 0, 0>>>(sampler_ptr);
#ifdef check
  print_result<<<1, 32, 0, 0>>>(sampler_ptr);
#endif
  HERR(hipDeviceSynchronize());
  HERR(hipPeekAtLastError());
}
