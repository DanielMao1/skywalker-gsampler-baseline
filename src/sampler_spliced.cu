#include "hip/hip_runtime.h"
#include "alias_table.cuh"
#include "sampler.cuh"
#include "util.cuh"
#define paster(n) printf("var: " #n " =  %d\n", n)

static __device__ void SampleWarpCentic(sample_result &result, gpu_graph *ggraph,
                                 hiprandState state, int current_itr, int idx,
                                 int node_id, void *buffer) {
  // __shared__ alias_table_constructor_shmem<uint, ExecutionPolicy::WC>
  // tables[WARP_PER_BLK];
  alias_table_constructor_shmem<uint, ExecutionPolicy::WC> *tables =
      (alias_table_constructor_shmem<uint, ExecutionPolicy::WC> *)buffer;
  alias_table_constructor_shmem<uint, ExecutionPolicy::WC> *table = &tables[WID];

  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero) {
    table->construct();
    table->roll_atomic(result.getNextAddr(current_itr), &state, result);
  }
  table->Clean();
}

static __device__ void SampleBlockCentic(sample_result &result, gpu_graph *ggraph,
                                  hiprandState state, int current_itr, int idx,
                                  int node_id, void *buffer,
                                  Buffer_pointer *buffer_pointer) {
  // __shared__ alias_table_constructor_shmem<uint, ExecutionPolicy::BC> tables[1];
  alias_table_constructor_shmem<uint, ExecutionPolicy::BC, BufferType::SPLICED> *tables =
      (alias_table_constructor_shmem<uint, ExecutionPolicy::BC, BufferType::SPLICED> *)
          buffer;
  alias_table_constructor_shmem<uint, ExecutionPolicy::BC, BufferType::SPLICED> *table =
      &tables[0];

#ifdef check
  if (LTID == 0)
    printf("GWID %d itr %d got one job idx %u node_id %u with degree %d \n ",
           GWID, current_itr, idx, node_id, ggraph->getDegree(node_id));
#endif
  if (ggraph->getDegree(node_id) > ELE_PER_BLOCK && buffer_pointer != nullptr)
    table->loadGlobalBuffer(buffer_pointer);
  __syncthreads();
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  __syncthreads();
  if (not_all_zero) {
    table->construct();
    uint target_size =
        MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    table->roll_atomic(result.getNextAddr(current_itr), target_size, &state,
                       result);
  }
  __syncthreads();
  table->Clean();
}

__global__ void sample_kernel(Sampler *sampler,
                              Buffer_pointer *buffer_pointers) {
  sample_result &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Buffer_pointer *buffer_pointer = &buffer_pointers[BID];

  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0)
    current_itr = 0;
  __syncthreads();
  // __shared__ char buffer[48928];
  __shared__ alias_table_constructor_shmem<uint, ExecutionPolicy::BC> table;
  void *buffer = &table;
  // void * buffer=nullptr;
  // __shared__ Vector_shmem<id_pair, ExecutionPolicy::BC, 32> high_degree_vec;
  Vector_gmem<uint> *high_degrees = &sampler->result.high_degrees[0];

  for (; current_itr < result.hop_num - 1;) {
    // TODO
    // high_degree_vec.Init(0);
    sample_job job;

    if (LID == 0)
      job = result.requireOneJob(current_itr);
    __syncwarp(0xffffffff);
    job.idx = __shfl_sync(0xffffffff, job.idx, 0);
    job.val = __shfl_sync(0xffffffff, job.val, 0);
    job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    while (job.val) {
      if (ggraph->getDegree(job.node_id) < ELE_PER_WARP) {
        SampleWarpCentic(result, ggraph, state, current_itr, job.idx,
                         job.node_id, buffer);
      } else {
        if (LID == 0) {
          // high_degree.idx = job.idx;
          // high_degree.node_id = job.node_id;
          // high_degree_vec.Add(high_degree);
          result.AddHighDegree(current_itr, job.node_id);
        }
      }
      __syncwarp(0xffffffff);
      if (LID == 0)
        job = result.requireOneJob(current_itr);
      job.idx = __shfl_sync(0xffffffff, job.idx, 0);
      job.val = __shfl_sync(0xffffffff, job.val, 0);
      job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    }
    __syncthreads();

    // for (size_t i = 0; i < high_degree_vec.Size(); i++)
    __shared__ sample_job high_degree_job;
    if (LTID == 0) {
      job = result.requireOneHighDegreeJob(current_itr);
      high_degree_job.val = job.val;
      high_degree_job.node_id = job.node_id;
    }
    __syncthreads();
    while (high_degree_job.val) {
      SampleBlockCentic(result, ggraph, state, current_itr, 0,
                        high_degree_job.node_id, buffer,
                        buffer_pointer); // buffer_pointer
      __syncthreads();
      if (LTID == 0) {
        job = result.requireOneHighDegreeJob(current_itr);
        high_degree_job.val = job.val;
        high_degree_job.node_id = job.node_id;
      }
      __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

static __global__ void init_kernel_ptr2(Sampler *sampler) {
  if (TID == 0) {
    sampler->result.setAddrOffset();
  }
}
static __global__ void print_result(Sampler *sampler) {
  sampler->result.PrintResult();
}

// void Start_high_degree(Sampler sampler)
void StartSP(Sampler sampler) {
  // orkut max degree 932101
  if (FLAGS_v)
    printf("%s:%d %s\n", __FILE__, __LINE__, __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  if (sizeof(alias_table_constructor_shmem<uint, ExecutionPolicy::BC>) <
      sizeof(alias_table_constructor_shmem<uint, ExecutionPolicy::WC>) * WARP_PER_BLK)
    printf("buffer too small\n");
  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  H_ERR(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                   hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr2<<<1, 32, 0, 0>>>(sampler_ptr);

  // allocate global buffer
  int block_num = n_sm * 1024 / BLOCK_SIZE;
  int gbuff_size = sampler.ggraph.MaxDegree;;
  LOG("alllocate GMEM buffer %d\n", block_num * gbuff_size * MEM_PER_ELE);
  Buffer_pointer *buffer_pointers = new Buffer_pointer[block_num];
  for (size_t i = 0; i < block_num; i++) {
    buffer_pointers[i].allocate(gbuff_size);
  }
  H_ERR(hipDeviceSynchronize());
  Buffer_pointer *buffer_pointers_g;
  H_ERR(hipMalloc(&buffer_pointers_g, sizeof(Buffer_pointer) * block_num));
  H_ERR(hipMemcpy(buffer_pointers_g, buffer_pointers,
                   sizeof(Buffer_pointer) * block_num, hipMemcpyHostToDevice));

  //  Global_buffer
  H_ERR(hipDeviceSynchronize());
  start_time = wtime();
#ifdef check
  sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, buffer_pointers_g);
#else
  sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr,
                                                 buffer_pointers_g);
#endif
  H_ERR(hipDeviceSynchronize());
  // H_ERR(hipPeekAtLastError());
  total_time = wtime() - start_time;
  printf("SamplingTime:\t%.6f\n", total_time);
  print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  H_ERR(hipDeviceSynchronize());
}
