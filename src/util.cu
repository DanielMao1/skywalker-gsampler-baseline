#include "hip/hip_runtime.h"
#include "util.cuh"

// __device__ char char_atomicCAS(char *addr, char cmp, char val) {
//   unsigned *al_addr = reinterpret_cast<unsigned *>(((unsigned long long)addr) &
//                                                    (0xFFFFFFFFFFFFFFFCULL));
//   unsigned al_offset = ((unsigned)(((unsigned long long)addr) & 3)) * 8;
//   unsigned mask = 0xFFU;
//   mask <<= al_offset;
//   mask = ~mask;
//   unsigned sval = val;
//   sval <<= al_offset;
//   unsigned old = *al_addr, assumed, setval;
//   do {
//     assumed = old;
//     setval = assumed & mask;
//     setval |= sval;
//     old = atomicCAS(al_addr, assumed, setval);
//   } while (assumed != old);
//   return (char)((assumed >> al_offset) & 0xFFU);
// }

// template <typename T>
// __inline__ __device__ T warpPrefixSum(T val, int lane_id) {
//   T val_shuffled;
//   for (int offset = 1; offset < warpSize; offset *= 2) {
//     val_shuffled = __shfl_up(val, offset);
//     if (lane_id >= offset) {
//       val += val_shuffled;
//     }
//   }
//   return val;
// }
double wtime()
{
  double time[2];
  struct timeval time1;
  gettimeofday(&time1, NULL);

  time[0] = time1.tv_sec;
  time[1] = time1.tv_usec;

  return time[0] + time[1] * 1.0e-6;
}
__device__ void __conv(){
  coalesced_group active = coalesced_threads();
}
__device__ void active_size(int n = 0)
{
  coalesced_group active = coalesced_threads();
  if (active.thread_rank() == 0)
    printf("WID: %d coalesced_group %llu at line %d\n", WID, active.size(), n);
}
template <typename T>
void printH(T *ptr, int size)
{
  T *ptrh = new T[size];
  HERR(hipMemcpy(ptrh, ptr, size * sizeof(T), hipMemcpyDeviceToHost));
  printf("printH: ");
  for (size_t i = 0; i < size; i++)
  {
    // printf("%d\t", ptrh[i]);
    std::cout << ptrh[i] << "\t";
  }
  printf("\n");
  delete ptrh;
}
__device__ void printD(float *ptr, int size)
{
  printf("printDf: size %d, ", size);
  for (size_t i = 0; i < size; i++)
  {
    printf("%f\t", ptr[i]);
  }
  printf("\n");
}
__device__ void printD(int *ptr, int size)
{
  printf("printDi: size %d, ", size);
  for (size_t i = 0; i < size; i++)
  {
    printf("%d\t", ptr[i]);
  }
  printf("\n");
}
__device__ void printD(uint32_t *ptr, int size)
{
  printf("printDi: size %d, ", size);
  for (size_t i = 0; i < size; i++)
  {
    printf("%u\t", ptr[i]);
  }
  printf("\n");
}
// template <typename T> __global__ void init_range_d(T *ptr, size_t size) {
//   if (TID < size) {
//     ptr[TID] = TID;
//   }
// }
// template <typename T> void init_range(T *ptr, size_t size) {
//   init_range_d<T><<<size / 512 + 1, 512>>>(ptr, size);
// }
// template <typename T> __global__ void init_array_d(T *ptr, size_t size, T v) {
//   if (TID < size) {
//     ptr[TID] = v;
//   }
// }
// template <typename T> void init_array(T *ptr, size_t size, T v) {
//   init_array_d<T><<<size / 512 + 1, 512>>>(ptr, size, v);
// }
