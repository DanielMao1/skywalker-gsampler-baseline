#include "hip/hip_runtime.h"
/*
 * @Description: just perform RW
 * @Date: 2020-11-30 14:30:06
 * @LastEditors: Please set LastEditors
 * @LastEditTime: 2021-01-15 15:58:34
 * @FilePath: /skywalker/src/unbiased_sample.cu
 */
#include "kernel.cuh"
#include "roller.cuh"
#include "sampler.cuh"
#include "util.cuh"
DECLARE_bool(v);
// DEFINE_bool(dynamic, false, "invoke kernel for each itr");
DECLARE_double(tp);
DECLARE_bool(printresult);

// #define paster(n) printf("var: " #n " =  %d\n", n)

// __global__ void UnbiasedWalkKernelPerItr(Sampler *sampler, uint current_itr)
// {
//   Jobs_result<JobType::RW, uint> &result = sampler->result;
//   gpu_graph *graph = &sampler->ggraph;
//   hiprandState state;
//   hiprand_init(TID, 0, 0, &state);
//   // for (uint current_itr = 0; current_itr < result.hop_num - 1;
//   current_itr++)
//   // {
//   if (TID < result.frontier.Size(current_itr)) {
//     size_t idx_i = result.frontier.Get(current_itr, TID);
//     uint src_id = result.GetData(current_itr, idx_i);
//     uint src_degree = graph->getDegree((uint)src_id);
//     if (1 < src_degree) {
//       int col = (int)floor(hiprand_uniform(&state) * src_degree);
//       uint candidate = col;
//       *result.GetDataPtr(current_itr + 1, idx_i) =
//           graph->getOutNode(src_id, candidate);
//       result.frontier.SetActive(current_itr + 1, idx_i);
//     } else {
//       *result.GetDataPtr(current_itr + 1, idx_i) = graph->getOutNode(src_id,
//       0); result.frontier.SetActive(current_itr + 1, idx_i);
//     }
//   }
//   // }
// }
// __global__ void Reset(Sampler *sampler, uint current_itr) {
//   if (TID == 0) sampler->result.frontier.Reset(current_itr);
// }
// __global__ void GetSize(Sampler *sampler, uint current_itr, uint *size) {
//   if (TID == 0) *size = sampler->result.frontier.Size(current_itr);
// }

// __global__ void UnbiasedSampleKernel(Sampler *sampler, float *tp) {
//   // Jobs_result<JobType::RW, uint> &result = sampler->result;
//   sample_result &result = sampler->result;
//   gpu_graph *graph = &sampler->ggraph;
//   hiprandState state;
//   hiprand_init(TID, 0, 0, &state);
//   for (size_t idx_i = TID; idx_i < result.size;
//        idx_i += gridDim.x * blockDim.x) {
//     result.length[idx_i] = result.hop_num - 1;
//     for (uint current_itr = 0; current_itr < result.hop_num - 1;
//          current_itr++) {
//       uint src_id = result.GetData(current_itr, idx_i);
//       uint src_degree = graph->getDegree((uint)src_id);
//       // if(idx_i==0) printf("src_id %d src_degree %d\n",src_id,src_degree );
//       if (src_degree == 0 || hiprand_uniform(&state) < *tp) {
//         result.length[idx_i] = current_itr;
//         break;
//       } else if (1 < src_degree) {
//         int col = (int)floor(hiprand_uniform(&state) * src_degree);
//         uint candidate = col;
//         *result.GetDataPtr(current_itr + 1, idx_i) =
//             graph->getOutNode(src_id, candidate);
//       } else {
//         *result.GetDataPtr(current_itr + 1, idx_i) =
//             graph->getOutNode(src_id, 0);
//       }
//     }
//   }
// }

static __global__ void sample_kernel(Sampler *sampler) {
  sample_result &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();

  for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
  {
    // if(LID==0) paster(result.hop_num - 1);
    sample_job job;
    __threadfence_block();
    // if (LID == 0)
    job = result.requireOneJob(current_itr);
    while (job.val ) { //&& graph->CheckValid(job.node_id)
      uint src_id = job.node_id;
      uint src_degree = graph->getDegree((uint)src_id);
      {
        uint target_size = result.hops[current_itr + 1];
        if ((target_size > 0) && (target_size < src_degree)) {
          //   int itr = 0;
          for (size_t i = 0; i < target_size; i++) {
            int col = (int)floor(hiprand_uniform(&state) * src_degree);
            float p = hiprand_uniform(&state);
            uint candidate = col;
            result.AddActive(current_itr, result.getNextAddr(current_itr),
                             graph->getOutNode(src_id, candidate));
          }
        } else if (target_size >= src_degree) {
          for (size_t i = 0; i < src_degree; i++) {
            result.AddActive(current_itr, result.getNextAddr(current_itr),
                             graph->getOutNode(src_id, i));
          }
        }
      }
      job = result.requireOneJob(current_itr);
    }
    __syncthreads();
    if (threadIdx.x == 0) result.NextItr(current_itr);
    __syncthreads();
  }
}

static __global__ void print_result(Sampler *sampler) {
  sampler->result.PrintResult();
}

float UnbiasedSample(Sampler &sampler) {
  LOG("%s\n", __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                          hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);

  // allocate global buffer
  int block_num = n_sm * 1024 / BLOCK_SIZE;
  CUDA_RT_CALL(hipDeviceSynchronize());
  CUDA_RT_CALL(hipPeekAtLastError());

  uint size_h, *size_d;
  hipMalloc(&size_d, sizeof(uint));

  start_time = wtime();
  sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr);
  // if (!FLAGS_dynamic) {
  //   UnbiasedSampleKernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, tp_d);
  // }
  // else {
  //   for (uint current_itr = 0; current_itr < sampler.result.hop_num - 1;
  //        current_itr++) {
  //     GetSize<<<1, 32, 0, 0>>>(sampler_ptr, current_itr, size_d);
  //     CUDA_RT_CALL(
  //         hipMemcpy(&size_h, size_d, sizeof(uint), hipMemcpyDeviceToHost));
  //     if (size_h > 0) {
  //       UnbiasedWalkKernelPerItr<<<size_h / BLOCK_SIZE + 1, BLOCK_SIZE, 0,
  //       0>>>(
  //           sampler_ptr, current_itr);
  //       Reset<<<1, 32, 0, 0>>>(sampler_ptr, current_itr);
  //     } else {
  //       break;
  //     }
  //   }
  // }

  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
  LOG("Device %d sampling time:\t%.6f ratio:\t %.2f MSEPS sampled %u\n",
      omp_get_thread_num(), total_time,
      static_cast<float>(sampler.result.GetSampledNumber() / total_time /
                         1000000),
      sampler.result.GetSampledNumber());
  sampler.sampled_edges = sampler.result.GetSampledNumber();
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
