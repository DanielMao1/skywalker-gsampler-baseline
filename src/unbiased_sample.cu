#include "hip/hip_runtime.h"
/*
 * @Description: just perform RW
 * @Date: 2020-11-30 14:30:06
 * @LastEditors: Pengyu Wang
 * @LastEditTime: 2022-03-03 19:55:50
 * @FilePath: /skywalker/src/unbiased_sample.cu
 */
#include "app.cuh"

static __global__ void sample_kernel_first(Sampler_new *sampler, uint itr) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  // __shared__ matrixBuffer<BLOCK_SIZE, 11, uint> buffer_1hop;
  // buffer_1hop.Init();
  size_t idx_i = TID;
  if (idx_i < result.size) {
    uint current_itr = 0;
    // coalesced_group active = coalesced_threads();
    {
      uint src_id = result.GetData(idx_i, current_itr, 0);
      uint src_degree = graph->getDegree((uint)src_id);
#ifdef UNIQUE_SAMPLE
      uint sample_size = MIN(src_degree, result.hops[current_itr + 1]);
      duplicate_checker<uint, 25> checker;
#else
      uint sample_size = result.hops[current_itr + 1];
#endif

      for (size_t i = 0; i < sample_size; i++) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
#ifdef UNIQUE_SAMPLE
        if (!checker.check(candidate))
          i--;
        else
#endif
          *result.GetDataPtr(idx_i, current_itr + 1, i) =
              graph->getOutNode(src_id, candidate);
        // if (!TID) printf("adding %u \n", graph->getOutNode(src_id,
        // candidate));
      }
      result.SetSampleLength(idx_i, current_itr, 0, sample_size);
    }
  }
}
static __global__ void sample_kernel_first_buffer(Sampler_new *sampler,
                                                  uint itr) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, 11, uint> buffer_1hop;
  buffer_1hop.Init();
  __syncthreads();
  size_t idx_i = TID;
  if (idx_i < result.size) {
    uint current_itr = 0;
    coalesced_group active = coalesced_threads();
    {
      uint src_id = result.GetData(idx_i, current_itr, 0);
      uint src_degree = graph->getDegree((uint)src_id);
#ifdef UNIQUE_SAMPLE
      uint sample_size = MIN(src_degree, result.hops[current_itr + 1]);
      duplicate_checker<uint, 25> checker;
#else
      uint sample_size = result.hops[current_itr + 1];
#endif

      for (size_t i = 0; i < sample_size; i++) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
        // *result.GetDataPtr(idx_i, current_itr + 1, i) =
        //       graph->getOutNode(src_id, candidate);
        // if (!idx_i)
        //   printf("adding %u \n", graph->getOutNode(src_id, candidate));
        buffer_1hop.Set(graph->getOutNode(src_id, candidate));
        buffer_1hop.CheckFlush(
            result.data + result.length_per_sample * idx_i - 1, current_itr,
            active);
      }
      active.sync();
      // if (!idx_i) printf("buffer_1hop.outItr %u \n", buffer_1hop.outItr[0]);
      buffer_1hop.Flush2(result.GetDataPtr(idx_i, 1, 0), active);
      // buffer_1hop.Flush(result.data + result.length_per_sample * idx_i, 0,
      //                   active);
      result.SetSampleLength(idx_i, current_itr, 0, sample_size);
    }
  }
}
template <uint subwarp_size>
static __global__ void sample_kernel_second(Sampler_new *sampler,
                                            uint current_itr) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  size_t subwarp_id = TID / subwarp_size;
  uint subwarp_idx = TID % subwarp_size;
  // uint local_subwarp_idx = LTID % subwarp_size;
  bool alive = (subwarp_idx < result.hops[current_itr]) ? 1 : 0;
  size_t idx_i = subwarp_id;  //

  if (idx_i < result.size)  // for 2-hop, hop_num=3
  {
    coalesced_group active = coalesced_threads();
    {
      uint src_id, src_degree, sample_size;
      if (alive) {
        src_id = result.GetData(idx_i, current_itr, subwarp_idx);
        src_degree = graph->getDegree((uint)src_id);
#ifdef UNIQUE_SAMPLE
        sample_size = MIN(src_degree, result.hops[current_itr + 1]);
        duplicate_checker<uint, 10> checker;
#else
        sample_size = result.hops[current_itr + 1];
#endif
        // if (!idx_i) printf("sample_size %u\n", sample_size);
        for (size_t i = 0; i < sample_size; i++) {
          uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
#ifdef UNIQUE_SAMPLE
          if (!checker.check(candidate))
            i--;
          else
#endif
          {
            // if (!idx_i && subwarp_idx == 1)
            //   printf("subwarp_idx 1 add %u\n", graph->getOutNode(src_id,
            //   candidate));
            *result.GetDataPtr(idx_i, current_itr + 1,
                               subwarp_idx * result.hops[2] + i) =
                graph->getOutNode(src_id, candidate);
          }
        }
      }
      if (alive)
        result.SetSampleLength(idx_i, current_itr, subwarp_idx, sample_size);
    }
  }
}
template <uint subwarp_size, uint buffer_size = 11>
static __global__ void sample_kernel_second_buffer(Sampler_new *sampler,
                                                   uint current_itr) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, buffer_size, uint> buffer;
  buffer.Init();
  size_t subwarp_id = TID / subwarp_size;
  uint subwarp_idx = TID % subwarp_size;
  bool alive = (subwarp_idx < result.hops[current_itr]) ? 1 : 0;
  size_t idx_i = subwarp_id;  //

  if (idx_i < result.size)  // for 2-hop, hop_num=3
  {
    coalesced_group active = coalesced_threads();
    {
      uint src_id, src_degree, sample_size;
      if (alive) {
        src_id = result.GetData(idx_i, current_itr, subwarp_idx);
        src_degree = graph->getDegree((uint)src_id);
#ifdef UNIQUE_SAMPLE
        sample_size = MIN(src_degree, result.hops[current_itr + 1]);
        duplicate_checker<uint, 10> checker;
#else
        sample_size = result.hops[current_itr + 1];
#endif
        // if (!idx_i) printf("sample_size %u\n", sample_size);
        for (size_t i = 0; i < sample_size; i++) {
          uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
#ifdef UNIQUE_SAMPLE
          if (!checker.check(candidate))
            i--;
          else
#endif
          {
            buffer.Set(graph->getOutNode(src_id, candidate));
            buffer.CheckFlush(result.GetDataPtr(idx_i, current_itr + 1,
                                                subwarp_idx * result.hops[2]) -
                                  1,
                              current_itr, active);
          }
        }
      }
      buffer.Flush2(result.GetDataPtr(idx_i, current_itr + 1,
                                      subwarp_idx * result.hops[2]),
                    active);
      if (alive)
        result.SetSampleLength(idx_i, current_itr, subwarp_idx, sample_size);
    }
  }
}

static __global__ void sample_kernel_2hop_buffer(Sampler_new *sampler) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, 10, uint> buffer_1hop;
  // __shared__ matrixBuffer<BLOCK_SIZE, 25, uint> buffer_2hop;  // not
  // necessary
  __shared__ uint idxMap[BLOCK_SIZE];
  idxMap[LTID] = 0;
  buffer_1hop.Init();
  // buffer_2hop.Init();

  size_t idx_i = TID;
  if (idx_i < result.size)  // for 2-hop, hop_num=3
  {
    idxMap[LTID] = idx_i;
    uint current_itr = 0;
    coalesced_group active = coalesced_threads();
    // 1-hop
    {
      uint src_id = result.GetData(idx_i, current_itr, 0);
      uint src_degree = graph->getDegree((uint)src_id);
#ifdef UNIQUE_SAMPLE
      uint sample_size = MIN(src_degree, result.hops[current_itr + 1]);
#else
      uint sample_size = result.hops[current_itr + 1];
#endif
      for (size_t i = 0; i < sample_size; i++) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
        buffer_1hop.Set(
            graph->getOutNode(src_id, candidate));  // can move back latter
      }
      active.sync();
      buffer_1hop.Flush(result.data + result.length_per_sample * idx_i, 0,
                        active);
      result.SetSampleLength(idx_i, current_itr, 0, sample_size);
    }
    current_itr = 1;
    // 2-hop  each warp for one???
    for (size_t i = 0; i < 32; i++) {  // loop over threads
      coalesced_group local = coalesced_threads();
      uint hop1_len;
      if (local.thread_rank() == 0) hop1_len = buffer_1hop.length[(WID)*32 + i];
      hop1_len = local.shfl(hop1_len, 0);

      for (size_t j = 0; j < MIN(result.hops[current_itr], hop1_len);
           j++) {  // loop over 1hop for each thread
        uint src_id =
            buffer_1hop.data[((WID)*32 + i) * buffer_1hop.tileLen + j];
        uint src_degree = graph->getDegree((uint)src_id);
#ifdef UNIQUE_SAMPLE
        uint sample_size = MIN(src_degree, result.hops[current_itr + 1]);
#else
        uint sample_size = result.hops[current_itr + 1];
#endif

        for (size_t k = active.thread_rank(); k < sample_size;
             k++) {  // get 2hop for 1hop neighbors for each thread
          uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
          *result.GetDataPtr(idxMap[(WID)*32 + i], current_itr + 1,
                             active.thread_rank()) =
              graph->getOutNode(src_id, candidate);
          // buffer_2hop.Set(graph->getOutNode(src_id, candidate));
        }
        // buffer_2hop.Flush(result.data + result.length_per_sample *
        // idxMap[(WID)*32 + i] + j*result.hops[current_itr] , 0);

        if (local.thread_rank() == 0) {
          result.SetSampleLength(idxMap[(WID)*32 + i], current_itr, j,
                                 sample_size);
        }
        local.sync();
      }
    }
  }
}

static __global__ void sample_kernel_2hop(Sampler_new *sampler) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  size_t idx_i = TID;
  if (idx_i < result.size)  // for 2-hop, hop_num=3
  {
    uint current_itr = 0;
    // 1-hop
    {
      uint src_id = result.GetData(idx_i, current_itr, 0);
      uint src_degree = graph->getDegree((uint)src_id);
      uint sample_size = MIN(result.hops[current_itr + 1], src_degree);
      for (size_t i = 0; i < sample_size; i++) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
        *result.GetDataPtr(idx_i, current_itr + 1, i) =
            graph->getOutNode(src_id, candidate);
        // if (!src_id)
        //   printf("add %d\t", graph->getOutNode(src_id, candidate));
      }
      // result.sample_lengths[idx_i*] = sample_size;
      result.SetSampleLength(idx_i, current_itr, 0, sample_size);
    }
    current_itr = 1;
    // 2-hop
    for (size_t k = 0; k < result.hops[current_itr]; k++) {
      uint src_id = result.GetData(idx_i, current_itr, k);
      uint src_degree = graph->getDegree((uint)src_id);
      uint sample_size = MIN(result.hops[current_itr + 1], src_degree);
      for (size_t i = 0; i < sample_size; i++) {
        uint candidate = (int)floor(hiprand_uniform(&state) * src_degree);
        *result.GetDataPtr(idx_i, current_itr + 1,
                           i + k * result.hops[current_itr + 1]) =
            graph->getOutNode(src_id, candidate);
        // if (!idx_i && k==1) printf("add %d\t", graph->getOutNode(src_id,
        // candidate));
      }
      // result.sample_lengths[idx_i*result.size_of_sample_lengths+ ] =
      // sample_size;
      result.SetSampleLength(idx_i, current_itr, k, sample_size);
    }
  }
}

static __global__ void print_result(Sampler_new *sampler) {
  sampler->result.PrintResult();
}

float UnbiasedSample(Sampler_new &sampler) {
              // double before_sample_start = wtime();

  // LOG("%s\n", __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;
  // LOG("overridding flags_itr for UnbiasedSample for better performance\n");
  FLAGS_peritr = 1;

  Sampler_new *sampler_ptr;
  MyCudaMalloc(&sampler_ptr, sizeof(Sampler_new));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler_new),
                          hipMemcpyHostToDevice));
  // double start_time, total_time;
  // init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr, false);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;  // 1024 / BLOCK_SIZE
  CUDA_RT_CALL(hipDeviceSynchronize());
  CUDA_RT_CALL(hipPeekAtLastError());

  uint size_h, *size_d;
  MyCudaMalloc(&size_d, sizeof(uint));

  // double before_sample_end = wtime();
  // printf("before sampler time:%.3f\n",(before_sample_end - before_sample_start) * 1000);
#pragma omp barrier
              double sample_start = wtime();
  if (FLAGS_peritr) {
    if (FLAGS_buffer) {
      sample_kernel_first_buffer<<<sampler.result.size / BLOCK_SIZE + 1,
                                   BLOCK_SIZE, 0, 0>>>(sampler_ptr, 0);
      CUDA_RT_CALL(hipDeviceSynchronize());
      sample_kernel_second_buffer<32, 11>
          <<<sampler.result.size * 32 / BLOCK_SIZE + 1, BLOCK_SIZE, 0, 0>>>(
              sampler_ptr, 1);
    } else {
      sample_kernel_first<<<sampler.result.size / BLOCK_SIZE + 1, BLOCK_SIZE, 0,
                            0>>>(sampler_ptr, 0);
      
      sample_kernel_second<32>
          <<<sampler.result.size * 32 / BLOCK_SIZE + 1, BLOCK_SIZE, 0, 0>>>(
              sampler_ptr, 1);
    }
  } else {
    if (FLAGS_buffer)
      sample_kernel_2hop_buffer<<<sampler.result.size / BLOCK_SIZE + 1,
                                  BLOCK_SIZE, 0, 0>>>(sampler_ptr);
    else
      sample_kernel_2hop<<<sampler.result.size / BLOCK_SIZE + 1, BLOCK_SIZE, 0,
                           0>>>(sampler_ptr);
  }

  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
              double sample_end = wtime();
              //   printf("sampler time:%.3f\n",(sample_end - sample_start) * 1000);
#pragma omp barrier
  // LOG("Device %d sampling time:\t%.6f ratio:\t %.2f MSEPS\n",
  //     omp_get_thread_num(), total_time,
  //     static_cast<float>(sampler.result.GetSampledNumber() / total_time /
  //                        1000000));
  sampler.sampled_edges = sampler.result.GetSampledNumber();
  MYLOG("sampled_edges %d\n", sampler.sampled_edges);
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
   if (sampler_ptr != nullptr) {
    CUDA_RT_CALL(hipFree(sampler_ptr));
    // printf("freed sampler!\n");
   }
  if (size_d != nullptr) {
    CUDA_RT_CALL(hipFree(size_d));
    // printf("freed sized_d!\n");
   }
  CUDA_RT_CALL(hipDeviceSynchronize());
                // double after_sample_end = wtime();
                // printf("after sampler time:%.3f\n",(after_sample_end - sample_end ) * 1000);
  return (float)(sample_end-sample_start);
}
