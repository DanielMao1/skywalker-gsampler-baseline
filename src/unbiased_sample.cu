#include "hip/hip_runtime.h"
/*
 * @Description: just perform RW
 * @Date: 2020-11-30 14:30:06
 * @LastEditors: Please set LastEditors
 * @LastEditTime: 2021-01-17 21:52:01
 * @FilePath: /skywalker/src/unbiased_sample.cu
 */
#include "app.cuh"

// __global__ void Reset(Sampler *sampler, uint current_itr) {
//   if (TID == 0) sampler->result.frontier.Reset(current_itr);
// }
// __global__ void GetSize(Sampler *sampler, uint current_itr, uint *size) {
//   if (TID == 0) *size = sampler->result.frontier.Size(current_itr);
// }

// __device__ void AsyncUnbiasedSampleJob(sample_result &result, gpu_graph *graph, uint instanceID, uint hop, uint offset){

// }
// __global__ void AsyncUnbiasedSampleKernel(Sampler *sampler, float *tp) {
//   // Jobs_result<JobType::RW, uint> &result = sampler->result;
//   sample_result &result = sampler->result;
//   gpu_graph *graph = &sampler->ggraph;
//   hiprandState state;
//   hiprand_init(TID, 0, 0, &state);
//   for (size_t idx_i = TID; idx_i < result.size;
//        idx_i += gridDim.x * blockDim.x) {
//     result.length[idx_i] = result.hop_num - 1;
//     for (uint current_itr = 0; current_itr < result.hop_num - 1;
//          current_itr++) {
//       uint src_id = result.getDataOfInstance(current_itr, idx_i);
//       uint src_degree = graph->getDegree((uint)src_id);
//       uint target_size = result.hops[current_itr + 1];
//       // if(idx_i==0) printf("src_id %d src_degree %d\n",src_id,src_degree );
//       if (src_degree == 0 || hiprand_uniform(&state) < *tp) {
//         // result.length[idx_i] = current_itr;
//         break;
//       } else {
//         if (target_size < src_degree) {
//           for (size_t i = 0; i < target_size; i++) {
//             int col = (int)floor(hiprand_uniform(&state) * src_degree);
//             uint candidate = col;
//             *(result.getAddrOfInstance(idx_i, current_itr + 1) + i) =
//                 graph->getOutNode(src_id, candidate);
//           }
//         } else {
//           for (size_t i = 0; i < src_degree; i++) {
//             *(result.getAddrOfInstance(idx_i, current_itr + 1) + i) =
//                 graph->getOutNode(src_id, i);
//           }
//         }
//       }
//     }
//   }
// }
static __global__ void SampleKernelPerItr(Sampler *sampler, uint current_itr) {
  sample_result &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  // __shared__ uint current_itr;
  // if (threadIdx.x == 0) current_itr = 0;
  // __syncthreads();

  // for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
  if (current_itr < result.hop_num - 1) {
    // if(LID==0) paster(result.hop_num - 1);
    sample_job job;
    __threadfence_block();
    // if (LID == 0)
    job = result.requireOneJob(current_itr);
    while (job.val) {  //&& graph->CheckValid(job.node_id)
      uint src_id = job.node_id;
      uint src_degree = graph->getDegree((uint)src_id);
      {
        uint target_size = result.hops[current_itr + 1];
        if ((target_size > 0) && (target_size < src_degree)) {
          //   int itr = 0;
          for (size_t i = 0; i < target_size; i++) {
            int col = (int)floor(hiprand_uniform(&state) * src_degree);
            float p = hiprand_uniform(&state);
            uint candidate = col;
            result.AddActive(current_itr, result.getNextAddr(current_itr),
                             graph->getOutNode(src_id, candidate));
          }
        } else if (target_size >= src_degree) {
          for (size_t i = 0; i < src_degree; i++) {
            result.AddActive(current_itr, result.getNextAddr(current_itr),
                             graph->getOutNode(src_id, i));
          }
        }
      }
      job = result.requireOneJob(current_itr);
    }
  }
}
static __global__ void sample_kernel(Sampler *sampler) {
  sample_result &result = sampler->result;
  gpu_graph *graph = &sampler->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();

  for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
  {
    // if(LID==0) paster(result.hop_num - 1);
    sample_job job;
    __threadfence_block();
    // if (LID == 0)
    job = result.requireOneJob(current_itr);
    while (job.val) {  //&& graph->CheckValid(job.node_id)
      uint src_id = job.node_id;
      uint src_degree = graph->getDegree((uint)src_id);
      {
        uint target_size = result.hops[current_itr + 1];
        if ((target_size > 0) && (target_size < src_degree)) {
          //   int itr = 0;
          for (size_t i = 0; i < target_size; i++) {
            int col = (int)floor(hiprand_uniform(&state) * src_degree);
            float p = hiprand_uniform(&state);
            uint candidate = col;
            result.AddActive(current_itr, result.getNextAddr(current_itr),
                             graph->getOutNode(src_id, candidate));
          }
        } else if (target_size >= src_degree) {
          for (size_t i = 0; i < src_degree; i++) {
            result.AddActive(current_itr, result.getNextAddr(current_itr),
                             graph->getOutNode(src_id, i));
          }
        }
      }
      job = result.requireOneJob(current_itr);
    }
    __syncthreads();
    if (threadIdx.x == 0) result.NextItr(current_itr);
    __syncthreads();
  }
}

static __global__ void print_result(Sampler *sampler) {
  sampler->result.PrintResult();
}
// __global__ void Reset(Sampler *walker, uint current_itr) {
//   if (TID == 0) walker->result.frontier.Reset(current_itr);
// }
__global__ void GetSize(Sampler *walker, uint current_itr, uint *size) {
  if (TID == 0) *size = walker->result.GetJobSize(current_itr);
}

float UnbiasedSample(Sampler &sampler) {
  LOG("%s\n", __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                          hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;  // 1024 / BLOCK_SIZE
  CUDA_RT_CALL(hipDeviceSynchronize());
  CUDA_RT_CALL(hipPeekAtLastError());

  uint size_h, *size_d;
  hipMalloc(&size_d, sizeof(uint));

  start_time = wtime();
  // if (true) {
  //   sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr);
  // } else {
  //   // to find the max occupation
  //   int dev = 0;
  //   int supportsCoopLaunch = 0;
  //   hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch,
  //                          dev);
  //   int numBlocksPerSm = 0;
  //   // Number of threads my_kernel will be launched with
  //   int numThreads = BLOCK_SIZE;
  //   hipDeviceProp_t deviceProp;
  //   hipGetDeviceProperties(&deviceProp, 0);
  //   hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,
  //                                                 sample_kernel, numThreads,
  //                                                 0);
  //   // launch
  //   void *kernelArgs[] = {sampler_ptr};
  //   dim3 dimBlock(numThreads, 1, 1);
  //   dim3 dimGrid(deviceProp.multiProcessorCount * numBlocksPerSm, 1, 1);
  //   // CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)sample_kernel,
  //   dimGrid,
  //   //                                          dimBlock, kernelArgs));
  //   sample_kernel<<<dimGrid, dimBlock, 0, 0>>>(sampler_ptr);
  // }
  if (!FLAGS_peritr) {
    sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr);
  } else {
    for (uint current_itr = 0; current_itr < sampler.result.hop_num - 1;
         current_itr++) {
      GetSize<<<1, 32, 0, 0>>>(sampler_ptr, current_itr, size_d);
      CUDA_RT_CALL(
          hipMemcpy(&size_h, size_d, sizeof(uint), hipMemcpyDeviceToHost));
      if (size_h > 0) {
        SampleKernelPerItr<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr,
                                                            current_itr);
        // Reset<<<1, 32, 0, 0>>>(sampler_ptr, current_itr);
      } else {
        break;
      }
    }
  }

  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
  LOG("Device %d sampling time:\t%.6f ratio:\t %.2f MSEPS sampled %u\n",
      omp_get_thread_num(), total_time,
      static_cast<float>(sampler.result.GetSampledNumber() / total_time /
                         1000000),
      sampler.result.GetSampledNumber());
  sampler.sampled_edges = sampler.result.GetSampledNumber();
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
