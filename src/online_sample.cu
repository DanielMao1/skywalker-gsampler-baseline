#include "hip/hip_runtime.h"
#include "alias_table.cuh"
#include "kernel.cuh"
#include "sampler.cuh"
#include "util.cuh"
#define paster(n) printf("var: " #n " =  %d\n", n)
DECLARE_bool(v);
static __device__ void SampleWarpCentic(sample_result &result, gpu_graph *ggraph,
                                 hiprandState state, int current_itr, int idx,
                                 int node_id, void *buffer) {
  alias_table_constructor_shmem<uint, ExecutionPolicy::WC> *tables =
      (alias_table_constructor_shmem<uint, ExecutionPolicy::WC> *)buffer;
  alias_table_constructor_shmem<uint, ExecutionPolicy::WC> *table =
      &tables[WID];
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero) {
    table->construct();
    table->roll_atomic(result.getNextAddr(current_itr), &state, result);
  }
  table->Clean();
}

static __device__ void SampleBlockCentic(sample_result &result, gpu_graph *ggraph,
                                  hiprandState state, int current_itr,
                                  int node_id, void *buffer,
                                  Vector_pack<uint> *vector_packs) {
  alias_table_constructor_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM>
      *tables = (alias_table_constructor_shmem<uint, ExecutionPolicy::BC,
                                               BufferType::GMEM> *)buffer;
  alias_table_constructor_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM>
      *table = &tables[0];
  table->loadGlobalBuffer(vector_packs);
  __syncthreads();
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  __syncthreads();
  if (not_all_zero) {
    table->constructBC();
    uint target_size =
        MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    table->roll_atomic(result.getNextAddr(current_itr), target_size, &state,
                       result);
  }
  __syncthreads();
  table->Clean();
}

__global__ void sample_kernel(Sampler *sampler,
                              Vector_pack<uint> *vector_pack) {
  sample_result &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  __shared__ alias_table_constructor_shmem<uint, ExecutionPolicy::WC>
      table[WARP_PER_BLK];
  void *buffer = &table[0];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0)
    current_itr = 0;
  __syncthreads();
  for (; current_itr < result.hop_num - 1;) // for 2-hop, hop_num=3
  {
    Vector_gmem<uint> *high_degrees = &sampler->result.high_degrees[current_itr];
    sample_job job;
    __threadfence_block();
    if (LID == 0)
      job = result.requireOneJob(current_itr);
    __syncwarp(0xffffffff);
    job.idx = __shfl_sync(0xffffffff, job.idx, 0);
    job.val = __shfl_sync(0xffffffff, job.val, 0);
    job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    __syncwarp(0xffffffff);
    while (job.val) {
      if (ggraph->getDegree(job.node_id) < ELE_PER_WARP) {
        SampleWarpCentic(result, ggraph, state, current_itr, job.idx,
                         job.node_id, buffer);
      } else {
        if (LID == 0)
          result.AddHighDegree(current_itr, job.node_id);
      }
      __syncwarp(0xffffffff);
      if (LID == 0)
        job = result.requireOneJob(current_itr);
      job.idx = __shfl_sync(0xffffffff, job.idx, 0);
      job.val = __shfl_sync(0xffffffff, job.val, 0);
      job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    }
    __syncthreads();
    __shared__ sample_job high_degree_job;
    if (LTID == 0) {
      job = result.requireOneHighDegreeJob(current_itr);
      high_degree_job.val = job.val;
      high_degree_job.node_id = job.node_id;
    }
    __syncthreads();
    while (high_degree_job.val) {
      SampleBlockCentic(result, ggraph, state, current_itr,
                        high_degree_job.node_id, buffer,
                        vector_packs); // buffer_pointer
      __syncthreads();
      if (LTID == 0) {
        job = result.requireOneHighDegreeJob(current_itr);
        high_degree_job.val = job.val;
        high_degree_job.node_id = job.node_id;
      }
      __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

static __global__ void print_result(Sampler *sampler) {
  sampler->result.PrintResult();
}

// void Start_high_degree(Sampler sampler)
void OnlineGBSample(Sampler sampler) {
  // orkut max degree 932101
  if (FLAGS_v)
    printf("%s:%d %s\n", __FILE__, __LINE__, __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  H_ERR(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                   hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);

  // allocate global buffer
  int block_num = n_sm * 1024 / BLOCK_SIZE;
  int gbuff_size = sampler.ggraph.MaxDegree;;
  LOG("alllocate GMEM buffer %d\n", block_num * gbuff_size * MEM_PER_ELE);

  Vector_pack<uint> *vector_pack_h = new Vector_pack<uint>[block_num];
  for (size_t i = 0; i < block_num; i++) {
    vector_pack_h[i].Allocate(gbuff_size);
  }
  H_ERR(hipDeviceSynchronize());
  Vector_pack<uint> *vector_packs;
  H_ERR(hipMalloc(&vector_packs, sizeof(Vector_pack<uint>) * block_num));
  H_ERR(hipMemcpy(vector_packs, vector_pack_h,
                   sizeof(Vector_pack<uint>) * block_num,
                   hipMemcpyHostToDevice));

  //  Global_buffer
  H_ERR(hipDeviceSynchronize());
  start_time = wtime();
#ifdef check
  sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#else
  sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#endif
  H_ERR(hipDeviceSynchronize());
  // H_ERR(hipPeekAtLastError());
  total_time = wtime() - start_time;
  printf("SamplingTime:\t%.6f\n", total_time);
  if(FLAGS_v)
  print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  H_ERR(hipDeviceSynchronize());
}
