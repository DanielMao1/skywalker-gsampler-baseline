#include "hip/hip_runtime.h"
#include "app.cuh"

template <>
__device__ bool
alias_table_constructor_shmem<uint, thread_block_tile<32>, BufferType::SHMEM>::
    roll_once<Jobs_result<JobType::NS, uint>>(
        uint *local_size, hiprandState *local_state, size_t target_size,
        Jobs_result<JobType::NS, uint> result, uint instance_id, uint offset,
        uint local_offset) {
  int col = (int)floor(hiprand_uniform(local_state) * buffer.size);
  float p = hiprand_uniform(local_state);
  uint candidate;
  if (p < buffer.prob[col])
    candidate = col;
  else
    candidate = buffer.alias[col];
  unsigned short int updated =
      atomicCAS(&buffer.selected[candidate], (unsigned short int)0,
                (unsigned short int)1);
  if (!updated) {
    auto active = coalesced_threads();
    // if (AddTillSize(local_size, target_size))
    if (*local_size + active.thread_rank() < target_size) {
      result.AddActive(buffer.current_itr + 1, instance_id, offset,
                       *local_size + active.thread_rank(),
                       buffer.ggraph->getOutNode(buffer.src_id, candidate),
                       (buffer.current_itr + 2) < result.hop_num);
    }
    if (active.thread_rank() == 0) {
      *local_size=MIN((*local_size+active.size()), target_size);
      // printf("1*local_size %u\n", *local_size);
    }

    return true;
  } else
    return false;
}
template <>
__device__ bool
alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM>::roll_once<
    Jobs_result<JobType::NS, uint>>(uint *local_size, hiprandState *local_state,
                                    size_t target_size,
                                    Jobs_result<JobType::NS, uint> result,
                                    uint instance_id, uint offset,
                                    uint local_offset) {
  int col = (int)floor(hiprand_uniform(local_state) * buffer.size);
  float p = hiprand_uniform(local_state);
  uint candidate;
  if (p < buffer.prob[col])
    candidate = col;
  else
    candidate = buffer.alias[col];
  unsigned short int updated =
      atomicCAS(&buffer.selected[candidate], (unsigned short int)0,
                (unsigned short int)1);
  if (!updated) {
    auto active = coalesced_threads();
    // if (AddTillSize(local_size, target_size))
    if (*local_size + active.thread_rank() < target_size) {
      result.AddActive(buffer.current_itr + 1, instance_id, offset,
                       *local_size + active.thread_rank(),
                       buffer.ggraph->getOutNode(buffer.src_id, candidate),
                       ((buffer.current_itr + 2) < result.hop_num));
    }
    if (active.thread_rank() == 0) {
      // *local_size += active.size();
      *local_size=MIN((*local_size+active.size()), target_size);
    }
    return true;
  } else
    return false;
}
template <>
__device__ void
alias_table_constructor_shmem<uint, thread_block_tile<32>, BufferType::SHMEM>::
    roll_atomic<Jobs_result<JobType::NS, uint>>(
        hiprandState *state, Jobs_result<JobType::NS, uint> result,
        uint instance_id, uint offset) {
  uint target_size = result.hops[buffer.current_itr + 1];
  // if (!LID)
  //   printf("src %u buffer.current_itr %u target_size %u\n", buffer.src_id,
  //          buffer.current_itr, target_size);
  if (target_size < buffer.ggraph->getDegree(buffer.src_id)) {
    int itr = 0;
    __shared__ uint sizes[WARP_PER_BLK];
    uint *local_size = sizes + WID;
    if (LID == 0) *local_size = 0;
    MySync();
    while (*local_size < target_size) {
      roll_once(local_size, state, target_size, result, instance_id, offset, 0);
      itr++;
      if (itr > 10) {
        break;
      }
      __syncwarp();
    }
    MySync();
  } else if (target_size >= buffer.ggraph->getDegree(buffer.src_id)) {
    target_size = buffer.ggraph->getDegree(buffer.src_id);
    for (size_t i = LID; i < target_size; i += 32) {
      result.AddActive(buffer.current_itr + 1, instance_id, offset, i,
                       buffer.ggraph->getOutNode(buffer.src_id, i),
                       (buffer.current_itr + 2) < result.hop_num);
    }
  }
  if (LID == 0) {
    result.SetSampleLength(instance_id, buffer.current_itr, offset,
                           target_size);
  }
  __syncwarp();
}

template <>
__device__ void
alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM>::
    roll_atomic<Jobs_result<JobType::NS, uint>>(
        int target_size, hiprandState *state,
        Jobs_result<JobType::NS, uint> result, uint instance_id, uint offset,
        uint local_offset) {
  __shared__ uint size;
  // if (!threadIdx.x)
  //   printf("src %u buffer.current_itr %u target_size %u\n", buffer.src_id,
  //          buffer.current_itr, target_size);
  // use only the first warp to sample
  if (WID == 0) {
    buffer.selected.CleanDataWC();
    int itr = 0;
    // uint *local_size = &sizes[0];
    if (LID == 0) size = 0;
    __syncwarp();
    while (size < target_size) {
      roll_once(&size, state, target_size, result, instance_id, offset, 0);
      itr++;
      if (itr > 10) {
        break;
      }
      __syncwarp();
    }
    if (LID == 0)
      result.SetSampleLength(instance_id, buffer.current_itr, offset, size);
  }
  __syncthreads_count(1);
}

static __device__ void SampleWarpCentic(Jobs_result<JobType::NS, uint> &result,
                                        gpu_graph *ggraph, hiprandState state,
                                        int current_itr, int instance_idx,
                                        int src_id, void *buffer, uint offset) {
  alias_table_constructor_shmem<uint, thread_block_tile<32>> *tables =
      (alias_table_constructor_shmem<uint, thread_block_tile<32>> *)buffer;
  alias_table_constructor_shmem<uint, thread_block_tile<32>> *table =
      &tables[WID];
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(src_id), ggraph,
                           ggraph->getDegree(src_id), current_itr, src_id);
  if (not_all_zero) {
    table->construct();
    table->roll_atomic(&state, result, instance_idx, offset);
  }
  table->Clean();
}

static __device__ void SampleBlockCentic(Jobs_result<JobType::NS, uint> &result,
                                         gpu_graph *ggraph, hiprandState state,
                                         int current_itr, int src_id,
                                         void *buffer,
                                         Vector_pack<uint> *vector_packs,
                                         uint instance_idx, uint offset) {
  alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM> *tables =
      (alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM> *)
          buffer;
  alias_table_constructor_shmem<uint, thread_block, BufferType::GMEM> *table =
      &tables[0];
  table->loadGlobalBuffer(vector_packs);
  __syncthreads_count(blockDim.x);
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(src_id), ggraph,
                           ggraph->getDegree(src_id), current_itr, src_id);
  __syncthreads_count(blockDim.x);
  if (not_all_zero) {
    table->constructBC();
    uint target_size =
        MIN(ggraph->getDegree(src_id), result.hops[current_itr + 1]);
    table->roll_atomic(target_size, &state, result, instance_idx, offset);
  }
  __syncthreads_count(blockDim.x);
  table->Clean();
}

#ifndef LOCALITY
__global__ void sample_kernel(Sampler_new *sampler,
                              Vector_pack<uint> *vector_pack) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  __shared__ alias_table_constructor_shmem<uint, thread_block_tile<32>>
      table[WARP_PER_BLK];
  void *buffer = &table[0];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();
  for (; current_itr < result.hop_num - 1;)  // for 2-hop, hop_num=3
  {
    // Vector_gmem<uint> *high_degrees =
    //     &sampler->result.high_degrees[current_itr];
    sampleJob<uint> job;
    __threadfence_block();
    if (LID == 0) job = result.requireOneJob(current_itr);
    __syncwarp(FULL_WARP_MASK);
    job.instance_idx = __shfl_sync(FULL_WARP_MASK, job.instance_idx, 0);
    job.offset = __shfl_sync(FULL_WARP_MASK, job.offset, 0);
    job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
    job.src_id = __shfl_sync(FULL_WARP_MASK, job.src_id, 0);
    __syncwarp(FULL_WARP_MASK);
    while (job.val) {
      if (ggraph->getDegree(job.src_id) < ELE_PER_WARP) {
        SampleWarpCentic(result, ggraph, state, current_itr, job.instance_idx,
                         job.src_id, buffer, job.offset);
      } else {
#ifdef skip8k
        if (LID == 0 && ggraph->getDegree(job.src_id) < 8000)
#else
        if (LID == 0)
#endif  // skip8k
          result.AddHighDegree(current_itr, job);
      }
      __syncwarp(FULL_WARP_MASK);
      if (LID == 0) job = result.requireOneJob(current_itr);
      __syncwarp(FULL_WARP_MASK);
      job.instance_idx = __shfl_sync(FULL_WARP_MASK, job.instance_idx, 0);
      job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
      job.src_id = __shfl_sync(FULL_WARP_MASK, job.src_id, 0);
      job.offset = __shfl_sync(FULL_WARP_MASK, job.offset, 0);
      // if (!LID) printf("%s:%d sync done  %d\n", __FILE__, __LINE__, WID);
    }
    __syncthreads();
    __shared__ sampleJob<uint> high_degree_job;
    if (LTID == 0) {
      job = result.requireOneHighDegreeJob(current_itr);
      high_degree_job.instance_idx = job.instance_idx;
      high_degree_job.val = job.val;
      high_degree_job.src_id = job.src_id;
      high_degree_job.offset = job.offset;
    }
    __syncthreads();
    while (high_degree_job.val) {
      SampleBlockCentic(result, ggraph, state, current_itr,
                        high_degree_job.src_id, buffer, vector_packs,
                        high_degree_job.instance_idx,
                        high_degree_job.offset);  // buffer_pointer
      __syncthreads();
      if (LTID == 0) {
        job = result.requireOneHighDegreeJob(current_itr);
        high_degree_job.instance_idx = job.instance_idx;
        high_degree_job.val = job.val;
        high_degree_job.src_id = job.src_id;
        high_degree_job.offset = job.offset;
      }
      __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      current_itr++;
    }
    __syncthreads();
  }
}
#else
__global__ void sample_kernel_loc(Sampler_new *sampler,
                                  Vector_pack<uint> *vector_pack) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  __shared__ alias_table_constructor_shmem<uint, thread_block_tile<32>>
      table[WARP_PER_BLK];

  void *buffer = &table[0];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  // __shared__ uint current_itr;
  // if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();
  while (result.frontier.needWork() || result.frontier.needWork()) {
    for (int current_bucket = 0; current_bucket < result.frontier._bucket_num;
         current_bucket++)  // for 2-hop, hop_num=3
    {
      while (result.frontier.checkFocus(current_bucket) ||
             result.high_degree.checkFocus(current_bucket)) {
        // Vector_gmem<uint> *high_degrees =
        //     &sampler->result.high_degrees[current_itr];
        sampleJob<uint> job;
        __threadfence_block();
        if (LID == 0)
          job = result.frontier.requireOneJobFromBucket(current_bucket);
        // {
        //   if (LID == 0 && (job.src_id == 430119 || job.src_id == 462435))
        //     printf(" got %u degree %d\n", job.src_id,
        //            ggraph->getDegree(job.src_id));
        // }
        __syncwarp(FULL_WARP_MASK);
        job.instance_idx = __shfl_sync(FULL_WARP_MASK, job.instance_idx, 0);
        job.offset = __shfl_sync(FULL_WARP_MASK, job.offset, 0);
        job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
        job.src_id = __shfl_sync(FULL_WARP_MASK, job.src_id, 0);
        job.itr = __shfl_sync(FULL_WARP_MASK, job.itr, 0);
        __syncwarp(FULL_WARP_MASK);
        while (job.val) {
          if (ggraph->getDegree(job.src_id) < ELE_PER_WARP) {
            SampleWarpCentic(result, ggraph, state, job.itr, job.instance_idx,
                             job.src_id, buffer, job.offset);
          } else {
#ifdef skip8k
            if (LID == 0 && ggraph->getDegree(job.src_id) < 8000)
#else
            if (LID == 0)
#endif  // skip8k
              result.AddHighDegree(job.itr, job);
          }
          // if (!LID) printf("%s:%d before  %d\n", __FILE__, __LINE__, WID);
          __syncwarp(FULL_WARP_MASK);
          // if (!LID) printf("%s:%d after  %d\n", __FILE__, __LINE__, WID);
          if (LID == 0)
            job = result.frontier.requireOneJobFromBucket(current_bucket);
          __syncwarp(FULL_WARP_MASK);
          job.instance_idx = __shfl_sync(FULL_WARP_MASK, job.instance_idx, 0);
          job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
          job.src_id = __shfl_sync(FULL_WARP_MASK, job.src_id, 0);
          job.offset = __shfl_sync(FULL_WARP_MASK, job.offset, 0);
          job.itr = __shfl_sync(FULL_WARP_MASK, job.itr, 0);
          // if (!LID) printf("%s:%d sync done  %d\n", __FILE__, __LINE__, WID);
        }
        __syncthreads();
        __shared__ sampleJob<uint> high_degree_job;
        if (LTID == 0) {
          job = result.high_degree.requireOneJobFromBucket(current_bucket);
          high_degree_job.instance_idx = job.instance_idx;
          high_degree_job.val = job.val;
          high_degree_job.src_id = job.src_id;
          high_degree_job.offset = job.offset;
          high_degree_job.itr = job.itr;
        }
        __syncthreads();
        while (high_degree_job.val) {
          SampleBlockCentic(result, ggraph, state, high_degree_job.itr,
                            high_degree_job.src_id, buffer, vector_packs,
                            high_degree_job.instance_idx,
                            high_degree_job.offset);  // buffer_pointer
          __syncthreads();
          if (LTID == 0) {
            job = result.high_degree.requireOneJobFromBucket(current_bucket);
            high_degree_job.instance_idx = job.instance_idx;
            high_degree_job.val = job.val;
            high_degree_job.src_id = job.src_id;
            high_degree_job.offset = job.offset;
            high_degree_job.itr = job.itr;
          }
          __syncthreads();
        }
        __syncthreads();
      }
    }
  }
}

__global__ void sample_kernel_loc2(Sampler_new *sampler,
                                   Vector_pack<uint> *vector_pack) {
  Jobs_result<JobType::NS, uint> &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  __shared__ alias_table_constructor_shmem<uint, thread_block_tile<32>>
      table[WARP_PER_BLK];

  void *buffer = &table[0];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  // __shared__ uint current_itr;
  // if (threadIdx.x == 0) current_itr = 0;
  __syncthreads();
  while (result.frontier.needWork() || result.frontier.needWork()) {
    for (int current_bucket = 0; current_bucket < result.frontier._bucket_num;
         current_bucket++)  // for 2-hop, hop_num=3
    {
      while (result.frontier.checkFocus(current_bucket) ||
             result.high_degree.checkFocus(current_bucket)) {
        __syncthreads();
        __shared__ sampleJob<uint> high_degree_job;
        sampleJob<uint> job;
        if (LTID == 0) {
          job = result.high_degree.requireOneJobFromBucket(current_bucket);
          high_degree_job.instance_idx = job.instance_idx;
          high_degree_job.val = job.val;
          high_degree_job.src_id = job.src_id;
          high_degree_job.offset = job.offset;
          high_degree_job.itr = job.itr;
        }
        __syncthreads();
        while (high_degree_job.val) {
          SampleBlockCentic(result, ggraph, state, high_degree_job.itr,
                            high_degree_job.src_id, buffer, vector_packs,
                            high_degree_job.instance_idx,
                            high_degree_job.offset);  // buffer_pointer
          __syncthreads();
          if (LTID == 0) {
            job = result.high_degree.requireOneJobFromBucket(current_bucket);
            high_degree_job.instance_idx = job.instance_idx;
            high_degree_job.val = job.val;
            high_degree_job.src_id = job.src_id;
            high_degree_job.offset = job.offset;
            high_degree_job.itr = job.itr;
          }
          __syncthreads();
        }

        if (LID == 0)
          job = result.frontier.requireOneJobFromBucket(current_bucket);
        __syncwarp(FULL_WARP_MASK);
        job.instance_idx = __shfl_sync(FULL_WARP_MASK, job.instance_idx, 0);
        job.offset = __shfl_sync(FULL_WARP_MASK, job.offset, 0);
        job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
        job.src_id = __shfl_sync(FULL_WARP_MASK, job.src_id, 0);
        job.itr = __shfl_sync(FULL_WARP_MASK, job.itr, 0);
        __syncwarp(FULL_WARP_MASK);
        while (job.val) {
          if (ggraph->getDegree(job.src_id) < ELE_PER_WARP) {
            SampleWarpCentic(result, ggraph, state, job.itr, job.instance_idx,
                             job.src_id, buffer, job.offset);
          } else {
            if (LID == 0) result.AddHighDegree(job.itr, job);
          }
          // if (!LID) printf("%s:%d before  %d\n", __FILE__, __LINE__, WID);
          __syncwarp(FULL_WARP_MASK);
          // if (!LID) printf("%s:%d after  %d\n", __FILE__, __LINE__, WID);
          if (LID == 0)
            job = result.frontier.requireOneJobFromBucket(current_bucket);
          __syncwarp(FULL_WARP_MASK);
          job.instance_idx = __shfl_sync(FULL_WARP_MASK, job.instance_idx, 0);
          job.val = __shfl_sync(FULL_WARP_MASK, job.val, 0);
          job.src_id = __shfl_sync(FULL_WARP_MASK, job.src_id, 0);
          job.offset = __shfl_sync(FULL_WARP_MASK, job.offset, 0);
          job.itr = __shfl_sync(FULL_WARP_MASK, job.itr, 0);
          // if (!LID) printf("%s:%d sync done  %d\n", __FILE__, __LINE__, WID);
        }
      }
    }
  }
}
#endif

static __global__ void print_result(Sampler_new *sampler) {
  sampler->result.PrintResult();
}

// void Start_high_degree(Sampler sampler)
float OnlineGBSampleNew(Sampler_new &sampler) {
  // orkut max degree 932101

  // LOG("%s\n", __FUNCTION__);
#ifdef skip8k
  LOG("skipping 8k\n");
#endif  // skip8k

  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Sampler_new *sampler_ptr;
  MyCudaMalloc(&sampler_ptr, sizeof(Sampler_new));
  CUDA_RT_CALL(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler_new),
                          hipMemcpyHostToDevice));
  double start_time, total_time;
  // init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr, true);

  int numBlocksPerSm = 0;
  // Number of threads my_kernel will be launched with
  int numThreads = BLOCK_SIZE;
  // hipOccupancyMaxActiveBlocksPerMultiprocessor(
  //     &numBlocksPerSm, sample_kernel, numThreads, 0);

  // paster(numBlocksPerSm);

  // allocate global buffer
  int block_num = n_sm * FLAGS_m;
#ifdef DEBUG
  block_num = 1;
#endif

  int gbuff_size = sampler.ggraph.MaxDegree + 10;

  LOG("alllocate GMEM buffer %d MB\n",
      block_num * gbuff_size * MEM_PER_ELE / 1024 / 1024);
  // paster(gbuff_size);
  Vector_pack<uint> *vector_pack_h = new Vector_pack<uint>[block_num];
  for (size_t i = 0; i < block_num; i++) {
    vector_pack_h[i].Allocate(gbuff_size, sampler.device_id);
  }
  CUDA_RT_CALL(hipDeviceSynchronize());
#pragma omp barrier
  Vector_pack<uint> *vector_packs;
  CUDA_RT_CALL(
      MyCudaMalloc(&vector_packs, sizeof(Vector_pack<uint>) * block_num));
  CUDA_RT_CALL(hipMemcpy(vector_packs, vector_pack_h,
                          sizeof(Vector_pack<uint>) * block_num,
                          hipMemcpyHostToDevice));

  //  Global_buffer
  CUDA_RT_CALL(hipDeviceSynchronize());
  start_time = wtime();
#ifndef NDEBUG
#ifdef LOCALITY
  {
    printf("%s:%d %s \n", __FILE__, __LINE__, "sample_kernel_loc");
    sample_kernel_loc<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
  }
#else
  {
    printf("%s:%d %s \n", __FILE__, __LINE__, "sample_kernel");
    sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
  }
#endif
#else
#ifdef LOCALITY
  sample_kernel_loc<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#else
  sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#endif
#endif
  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
#pragma omp barrier
  sampler.sampled_edges = sampler.result.GetSampledNumber(!FLAGS_peritr);
  LOG("Device %d sampling time:\t%.2f ms ratio:\t %.1f MSEPS\n",
      omp_get_thread_num(), total_time * 1000,
      static_cast<float>(sampler.sampled_edges / total_time / 1000000));
  LOG("sampled_edges %d\n", sampler.sampled_edges);
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  // sampler.result.printSize();
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
