#include "hip/hip_runtime.h"
#include "alias_table.cuh"
#include "sampler.cuh"
#include "util.cuh"
#define paster(n) printf("var: " #n " =  %d\n", n)

struct id_pair
{
  uint idx, node_id;
  __device__ id_pair &operator=(uint idx)
  {
    idx = 0;
    node_id = 0;
    return *this;
  }
};

__device__ void SampleWarpCentic(sample_result &result, gpu_graph *ggraph,
                                 hiprandState state, int current_itr, int idx,
                                 int node_id, void *buffer)
{
  // __shared__ alias_table_shmem<uint, ExecutionPolicy::WC>
  // tables[WARP_PER_SM];
  alias_table_shmem<uint, ExecutionPolicy::WC> *tables =
      (alias_table_shmem<uint, ExecutionPolicy::WC> *)buffer;
  alias_table_shmem<uint, ExecutionPolicy::WC> *table = &tables[WID];

  // #ifdef check
  //   if (LID == 0)
  //     printf("GWID %d itr %d got one job idx %u node_id %u with degree %d
  //     \n",
  //            GWID, current_itr, idx, node_id, ggraph->getDegree(node_id));
  // #endif
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  if (not_all_zero)
  {
    table->construct();
    uint target_size =
        MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    table->roll_atomic(result.getNextAddr(current_itr), target_size, &state,
                       result);
  }
  table->Clean();
}

__device__ void SampleBlockCentic(sample_result &result, gpu_graph *ggraph,
                                  hiprandState state, int current_itr, int idx,
                                  int node_id, void *buffer,
                                  Vector_pack<uint> *vector_packs)
{
  // __shared__ alias_table_shmem<uint, ExecutionPolicy::BC> tables[1];
  alias_table_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM> *tables =
      (alias_table_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM> *)buffer;
  alias_table_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM> *table = &tables[0];

#ifdef check
  if (LTID == 0)
    printf("GWID %d itr %d got one job idx %u node_id %u with degree %d \n ",
           GWID, current_itr, idx, node_id, ggraph->getDegree(node_id));
#endif

  // if (ggraph->getDegree(node_id) > ELE_PER_BLOCK && vector_packs != nullptr)
  table->loadGlobalBuffer(vector_packs);
  __syncthreads();
  bool not_all_zero =
      table->loadFromGraph(ggraph->getNeighborPtr(node_id), ggraph,
                           ggraph->getDegree(node_id), current_itr, node_id);
  __syncthreads();
  if (not_all_zero)
  {
    table->construct();
    uint target_size =
        MIN(ggraph->getDegree(node_id), result.hops[current_itr + 1]);
    table->roll_atomic(result.getNextAddr(current_itr), target_size, &state,
                       result);
  }
  __syncthreads();
  table->Clean();
}

__global__ void sample_kernel(Sampler *sampler,
                              Vector_pack<uint> *vector_pack)
{
  sample_result &result = sampler->result;
  gpu_graph *ggraph = &sampler->ggraph;
  Vector_pack<uint> *vector_packs = &vector_pack[BID];
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  __shared__ uint current_itr;
  if (threadIdx.x == 0)
    current_itr = 0;
  __syncthreads();
  // __shared__ char buffer[48928];
  __shared__ alias_table_shmem<uint, ExecutionPolicy::BC> table;
  void *buffer = &table;

  __shared__ Vector_shmem<id_pair, ExecutionPolicy::BC, 16> high_degree_vec;

  for (; current_itr < result.hop_num - 1;)
  {
    // TODO
    high_degree_vec.Init(0);

    id_pair high_degree;

    sample_job job;

    if (LID == 0)
      job = result.requireOneJob(current_itr);
    __syncwarp(0xffffffff);
    job.idx = __shfl_sync(0xffffffff, job.idx, 0);
    job.val = __shfl_sync(0xffffffff, job.val, 0);
    job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
    if (job.val)
    {
      if (ggraph->getDegree(job.node_id) < ELE_PER_WARP)
      {
        SampleWarpCentic(result, ggraph, state, current_itr, job.idx,
                         job.node_id, buffer);
        if (LID == 0)
          job = result.requireOneJob(current_itr);
        job.idx = __shfl_sync(0xffffffff, job.idx, 0);
        job.val = __shfl_sync(0xffffffff, job.val, 0);
        job.node_id = __shfl_sync(0xffffffff, job.node_id, 0);
      }
      else
      {
        if (LID == 0)
        {
          high_degree.idx = job.idx;
          high_degree.node_id = job.node_id;
          high_degree_vec.Add(high_degree);
        }
        __syncwarp(0xffffffff);
      }
    }
    __syncthreads();

    for (size_t i = 0; i < high_degree_vec.Size(); i++)
    {
      SampleBlockCentic(result, ggraph, state, current_itr,
                        high_degree_vec[i].idx, high_degree_vec[i].node_id,
                        buffer, vector_packs); // vector_packs
    }
    __syncthreads();
    if (threadIdx.x == 0)
    {
      result.NextItr(current_itr);
    }
    __syncthreads();
  }
}

__global__ void init_kernel_ptr(Sampler *sampler)
{
  if (TID == 0)
  {
    sampler->result.setAddrOffset();
  }
}
__global__ void print_result(Sampler *sampler)
{
  sampler->result.PrintResult();
}

// void Start_high_degree(Sampler sampler)
void Start(Sampler sampler)
{
  // orkut max degree 932101

  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  // paster( sizeof(alias_table_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM> ) );
  // paster( sizeof(alias_table_shmem<uint, ExecutionPolicy::WC>) * WARP_PER_SM );

  if (sizeof(alias_table_shmem<uint, ExecutionPolicy::BC, BufferType::GMEM>) <
      sizeof(alias_table_shmem<uint, ExecutionPolicy::WC>) * WARP_PER_SM)
    printf("buffer too small\n");
  Sampler *sampler_ptr;
  hipMalloc(&sampler_ptr, sizeof(Sampler));
  H_ERR(hipMemcpy(sampler_ptr, &sampler, sizeof(Sampler),
                   hipMemcpyHostToDevice));
  double start_time, total_time;
  init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr);

  // allocate global buffer
  Vector_pack<uint> *vector_pack_h = new Vector_pack<uint>[n_sm];
  for (size_t i = 0; i < n_sm; i++)
  {
    vector_pack_h[i].Allocate(932101);
  }
  HERR(hipDeviceSynchronize());
  Vector_pack<uint> *vector_packs;
  H_ERR(hipMalloc(&vector_packs, sizeof(Vector_pack<uint>) * n_sm));
  H_ERR(hipMemcpy(vector_packs, vector_pack_h,
                   sizeof(Vector_pack<uint>) * n_sm, hipMemcpyHostToDevice));

  //  Global_buffer

  start_time = wtime();
#ifdef check
  sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#else
  sample_kernel<<<n_sm, BLOCK_SIZE, 0, 0>>>(sampler_ptr, vector_packs);
#endif
  total_time = wtime() - start_time;
  printf("SamplingTime:%.6f\n", total_time);
  print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  HERR(hipDeviceSynchronize());
  HERR(hipPeekAtLastError());
}
